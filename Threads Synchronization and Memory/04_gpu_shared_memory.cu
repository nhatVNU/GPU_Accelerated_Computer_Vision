#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>

#define N   10

__global__ void gpu_shared_memory(float *d_in)
{
    int i;
    int index = threadIdx.x;
    float average;
    float sum = 0.0f;
    /* Defining shared memory */
    __shared__ float share_array[N];
    share_array[index] = d_in[index];

    /* This directive ensure all the writes to share memory have completed */
    __syncthreads();
    for (i = 0; i <= index; i++) {
        sum += share_array[i];
    }
    average = sum / (index + 1.0f);
    d_in[index] = average;
    share_array[index] = average;
}

int main() 
{
    float h_in[10];
    float* d_in;

    for (int i = 0; i < 10; i++) {
        h_in[i] = i;
    }

    hipMalloc((void**)&d_in, 10 * sizeof(float));
    hipMemcpy(d_in, h_in, 10 * sizeof(float), hipMemcpyHostToDevice);
    gpu_shared_memory << <1, 10 >> > (d_in);
    hipMemcpy(h_in, d_in, 10 * sizeof(float), hipMemcpyDeviceToHost);
    printf("Use of shared memory on GPU: \n");
    for (int i = 0; i < 10; i++)
    {
        printf("The running average after %d element is %f \n", i, h_in[i]);
    }
    return 0;
}