#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>

#define NUM_THREADS     10000
#define SIZE            10
#define BLOCK_WIDTH     100

__global__ void gpu_increment_without_atomic(int *d_in)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    tid = tid % SIZE;
    //d_in[tid] += 1;
    atomicAdd(&d_in[tid], 1);
}

int main() 
{
    printf("%d total threads in %d blocks writing into %d array elements\n", NUM_THREADS, NUM_THREADS / BLOCK_WIDTH, SIZE);

    int h_in[SIZE];
    const int ARRAY_BYTES = SIZE * sizeof(int);
    int* d_in;
    hipMalloc((void**)&d_in, ARRAY_BYTES);

    hipMemset((void*)d_in, 0, ARRAY_BYTES);
    gpu_increment_without_atomic << <NUM_THREADS / BLOCK_WIDTH, BLOCK_WIDTH >> > (d_in);
    hipMemcpy(h_in, d_in, ARRAY_BYTES, hipMemcpyDeviceToHost);

    printf("Number of times a particular Array index has been incremented without atomic add is: \n");
    for (int i = 0; i < SIZE; i++)
    {
        printf("index: %d --> %d times\n ", i, h_in[i]);
    }
    hipFree(d_in);
    return 0;
}