#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>

#define N 5

__global__ void gpu_local_memory(int d_in)
{
	int t_local;    
	t_local = d_in * threadIdx.x;     
	printf("Value of Local variable in current thread is: %d \n", t_local);
}

int main(int argc, char **argv)
{

	printf("Use of Local Memory on GPU:\n");
	gpu_local_memory << <1, N >> >(5);  
	hipDeviceSynchronize();
	return 0;
}